#include <string>
#include <chrono>

#define NDEBUG 1

#include <prover_reference_functions.hpp>

#include "multiexp/reduce_wnaf.cu"

#define ENABLE_WNAF

// This is where all the FFTs happen

// template over the bundle of types and functions.
// Overwrites ca!
template <typename B>
typename B::vector_Fr *compute_H(size_t d, typename B::vector_Fr *ca,
                                 typename B::vector_Fr *cb,
                                 typename B::vector_Fr *cc) {
  auto domain = B::get_evaluation_domain(d + 1);

  B::domain_iFFT(domain, ca);
  B::domain_iFFT(domain, cb);

  B::domain_cosetFFT(domain, ca);
  B::domain_cosetFFT(domain, cb);

  // Use ca to store H
  auto H_tmp = ca;

  size_t m = B::domain_get_m(domain);
  // for i in 0 to m: H_tmp[i] *= cb[i]
  B::vector_Fr_muleq(H_tmp, cb, m);

  B::domain_iFFT(domain, cc);
  B::domain_cosetFFT(domain, cc);

  m = B::domain_get_m(domain);

  // for i in 0 to m: H_tmp[i] -= cc[i]
  B::vector_Fr_subeq(H_tmp, cc, m);

  B::domain_divide_by_Z_on_coset(domain, H_tmp);

  B::domain_icosetFFT(domain, H_tmp);

  m = B::domain_get_m(domain);
  typename B::vector_Fr *H_res = B::vector_Fr_zeros(m + 1);
  B::vector_Fr_copy_into(H_tmp, H_res, m);
  return H_res;
}

static size_t read_size_t(FILE* input) {
  size_t n;
  fread((void *) &n, sizeof(size_t), 1, input);
  return n;
}

template< typename B >
struct ec_type;

template<>
struct ec_type<mnt4753_libsnark> {
    typedef ECp_MNT4 ECp;
    typedef ECp2_MNT4 ECpe;
};

template<>
struct ec_type<mnt6753_libsnark> {
    typedef ECp_MNT6 ECp;
    typedef ECp3_MNT6 ECpe;
};


void
check_trailing(FILE *f, const char *name) {
    long bytes_remaining = 0;
    while (fgetc(f) != EOF)
        ++bytes_remaining;
    if (bytes_remaining > 0)
        fprintf(stderr, "!! Trailing characters in \"%s\": %ld\n", name, bytes_remaining);
}


static inline auto now() -> decltype(std::chrono::high_resolution_clock::now()) {
    return std::chrono::high_resolution_clock::now();
}

template<typename T>
void
print_time(T &t1, const char *str) {
    auto t2 = std::chrono::high_resolution_clock::now();
    auto tim = std::chrono::duration_cast<std::chrono::milliseconds>(t2 - t1).count();
    printf("%s: %ld ms\n", str, tim);
    t1 = t2;
}

template <typename B>
void run_prover(
        const char *params_path,
        const char *input_path,
        const char *output_path,
        const char *preprocessed_path)
{
    B::init_public_params();

    size_t primary_input_size = 1;

    auto beginning = now();
    auto t = beginning;

    FILE *params_file = fopen(params_path, "r");
    size_t d = read_size_t(params_file);
    size_t m = read_size_t(params_file);
    rewind(params_file);

    printf("d = %zu, m = %zu\n", d, m);

    typedef typename ec_type<B>::ECp ECp;
    typedef typename ec_type<B>::ECpe ECpe;

    typedef typename B::G1 G1;
    typedef typename B::G2 G2;

    static constexpr int R = 32;
    static constexpr int C = 5;
    FILE *preprocessed_file = fopen(preprocessed_path, "r");

    size_t space = ((m + 1) + R - 1) / R;

    //auto A_mults = load_points_affine<ECp>(((1U << C) - 1)*(m + 1), preprocessed_file);
    //auto out_A = allocate_memory(space * ECpe::NELTS * ELT_BYTES);

#ifdef ENABLE_WNAF
    auto B1_mults = load_points_affine<ECp>(((1U << C))*(m + 1), preprocessed_file);
    auto B2_mults = load_points_affine<ECpe>(((1U << C))*(m + 1), preprocessed_file);
    auto L_mults = load_points_affine<ECp>(((1U << C))*(m - 1), preprocessed_file);
#else
    auto B1_mults = load_points_affine<ECp>(((1U << C) - 1)*(m + 1), preprocessed_file);
    auto B2_mults = load_points_affine<ECpe>(((1U << C) - 1)*(m + 1), preprocessed_file);
    auto L_mults = load_points_affine<ECp>(((1U << C) - 1)*(m - 1), preprocessed_file);
#endif
    auto out_B1 = allocate_memory(space * ECpe::NELTS * ELT_BYTES);
    auto out_B2 = allocate_memory(space * ECpe::NELTS * ELT_BYTES);
    auto out_L = allocate_memory(space * ECpe::NELTS * ELT_BYTES);

    fclose(preprocessed_file);

#if 0
    auto data = (uint16_t *)B1_mults.get();
    for (size_t i = 3600; i < 3840; i++) {
        if (i > 0 && i % 8 == 0) {
            printf("\n");
        }
        printf("%04x ", data[i]);
    }
    printf("\n");
#endif

    print_time(t, "load preprocessing");

    auto params = B::read_params(params_file, d, m);
    fclose(params_file);
    print_time(t, "load params");

    auto t_main = t;

    FILE *inputs_file = fopen(input_path, "r");
    auto w_ = load_scalars(m + 1, inputs_file);
    rewind(inputs_file);
    auto inputs = B::read_input(inputs_file, d, m);
    fclose(inputs_file);
    print_time(t, "load inputs");

    const var *w = w_.get();

#ifdef ENABLE_WNAF
    std::vector<long> wnafL;
    B::calc_wnaf(wnafL, B::input_w(inputs), m + 1);
    const size_t WNAF_SIZE = 769;
#if 0
    for (size_t i = 0; i < m + 1; i++) {
        printf("[Scalar #%d]:\n", i);
        B::print_Fr(B::input_w(inputs), i);
        for (size_t j = 0; j < WNAF_SIZE; j++) {
            printf("%d,", wnafL[i*WNAF_SIZE + j]);
        }
        printf("\n");
    }
#endif
    auto wnaf = alloc_memory<int8_t>(WNAF_SIZE * (m + 1));
    for (size_t i = 0; i < wnafL.size(); i++) {
        wnaf[i] = (int8_t)wnafL[i];
#if 0
        printf("%d,", wnaf[i]);
        if (i > 0 && i % (WNAF_SIZE-1) == 0) printf("\n");
#endif
    }

    print_time(t, "wnaf prepared");
#endif

    auto t_gpu = t;

    hipStream_t sA, sB1, sB2, sL;

#ifdef ENABLE_WNAF
    ec_reduce_wnaf<ECp, C, R>(sB1, out_B1.get(), B1_mults.get(), wnaf, m + 1);
    ec_reduce_wnaf<ECpe, C, 2*R>(sB2, out_B2.get(), B2_mults.get(), wnaf, m + 1);
    ec_reduce_wnaf<ECp, C, R>(sL, out_L.get(), L_mults.get(), wnaf + (primary_input_size + 1) * WNAF_SIZE, m - 1);
#else
    //ec_reduce_straus<ECp, C, R>(sA, out_A.get(), A_mults.get(), w, m + 1);
    ec_reduce_straus<ECp, C, R>(sB1, out_B1.get(), B1_mults.get(), w, m + 1);
    ec_reduce_straus<ECpe, C, 2*R>(sB2, out_B2.get(), B2_mults.get(), w, m + 1);
    ec_reduce_straus<ECp, C, R>(sL, out_L.get(), L_mults.get(), w + (primary_input_size + 1) * ELT_LIMBS, m - 1);
#endif
    print_time(t, "gpu launch");

    G1 *evaluation_At = B::multiexp_G1(B::input_w(inputs), B::params_A(params), m + 1);
    //G1 *evaluation_Bt1_cpu = B::multiexp_G1(B::input_w(inputs), B::params_B1(params), m + 1);
    //G2 *evaluation_Bt2 = B::multiexp_G2(B::input_w(inputs), B::params_B2(params), m + 1);

    // Do calculations relating to H on CPU after having set the GPU in
    // motion
    auto H = B::params_H(params);
    auto coefficients_for_H =
        compute_H<B>(d, B::input_ca(inputs), B::input_cb(inputs), B::input_cc(inputs));
    G1 *evaluation_Ht = B::multiexp_G1(coefficients_for_H, H, d);

    print_time(t, "cpu 1");

    hipDeviceSynchronize();
    //hipStreamSynchronize(sA);
    //G1 *evaluation_At = B::read_pt_ECp(out_A.get());

    hipStreamSynchronize(sB1);
    G1 *evaluation_Bt1 = B::read_pt_ECp(out_B1.get());

#if 0
    printf("#####CPU#####\n");
    B::print_G1(evaluation_Bt1_cpu);
    printf("#####GPU#####\n");
    B::print_G1(evaluation_Bt1);
    printf("#####GPU wnaf#####\n");
    B::print_G1(evaluation_Bt1_wnaf);
#endif

    hipStreamSynchronize(sB2);
    G2 *evaluation_Bt2 = B::read_pt_ECpe(out_B2.get());

    hipStreamSynchronize(sL);
    G1 *evaluation_Lt = B::read_pt_ECp(out_L.get());

    print_time(t_gpu, "gpu e2e");

    auto scaled_Bt1 = B::G1_scale(B::input_r(inputs), evaluation_Bt1);
    auto Lt1_plus_scaled_Bt1 = B::G1_add(evaluation_Lt, scaled_Bt1);
    auto final_C = B::G1_add(evaluation_Ht, Lt1_plus_scaled_Bt1);

    print_time(t, "cpu 2");

    B::groth16_output_write(evaluation_At, evaluation_Bt2, final_C, output_path);

    print_time(t, "store");

    print_time(t_main, "Total time from input to output: ");

    //hipStreamDestroy(sA);
    hipStreamDestroy(sB1);
    hipStreamDestroy(sB2);
    hipStreamDestroy(sL);

    B::delete_vector_G1(H);

    B::delete_G1(evaluation_At);
    B::delete_G1(evaluation_Bt1);
    B::delete_G2(evaluation_Bt2);
    B::delete_G1(evaluation_Ht);
    B::delete_G1(evaluation_Lt);
    B::delete_G1(scaled_Bt1);
    B::delete_G1(Lt1_plus_scaled_Bt1);
    B::delete_vector_Fr(coefficients_for_H);
    B::delete_groth16_input(inputs);
    B::delete_groth16_params(params);

    print_time(t, "cleanup");
    print_time(beginning, "Total runtime (incl. file reads)");
}

int main(int argc, char **argv) {
  setbuf(stdout, NULL);
  std::string curve(argv[1]);
  std::string mode(argv[2]);

  const char *params_path = argv[3];

  if (mode == "compute") {
      const char *input_path = argv[4];
      const char *output_path = argv[5];

      if (curve == "MNT4753") {
          run_prover<mnt4753_libsnark>(params_path, input_path, output_path, "MNT4753_preprocessed");
      } else if (curve == "MNT6753") {
          run_prover<mnt6753_libsnark>(params_path, input_path, output_path, "MNT6753_preprocessed");
      }
  } else if (mode == "preprocess") {
#if 0
      if (curve == "MNT4753") {
          run_preprocess<mnt4753_libsnark>(params_path);
      } else if (curve == "MNT6753") {
          run_preprocess<mnt4753_libsnark>(params_path);
      }
#endif
  }

  return 0;
}
